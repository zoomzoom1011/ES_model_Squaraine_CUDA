#include "hip/hip_runtime.h"
//

// the way to code the magma 
// 1. define all the objects in the CUDA and magma
// 2. allocate GPU memory for the matrix (hipMalloc)
// 3. copy the matrix to the the GPU
// 4. use diagonalization function in magma
// 5. copy the result from GPU to Memory
// 6. delete CUDA & magma objects and GPU memory space

#include "esm_c_magma.h" 


// Global variable to catch interrupt and terminate signals
volatile sig_atomic_t interrupted=false;


//use input name to get output name
string FileName(string name)
{
    int site = name.find_last_of('.');
    if (site == 0)
    {
        name = "output";
    }
    else if (site > 0)
    {
        name.erase(site, name.size() - site + 1);
    }
    return name;
}


//host name function
string HostName()
{
    string hostname;
    //for linux system
    ifstream hostname_file("/etc/hostname", ifstream::in);
    hostname_file >> hostname;
    return hostname;
}

// void mkdir(string dir)
// {
    // ofstream file("mkdir.bat");
    // file << "@echo off" << endl;
    // file << "if not exist " << dir << " mkdir " << dir << endl;
    // file << "chdir " << dir << endl;
    // file.close();

    // system("mkdir.bat");

    // remove("mkdir.bat");

    // cout << "**************************************************************" << endl;
    // cout << "entering ./" << dir << endl;
// }

int get_mon_state(int sysnx, int n) {
    int temp = (sysnx - 1) / pow(mon_kount, (n - 1));
    int get_mon = temp % mon_kount + 1;
    return get_mon;
}

int get_numvib(int molecule_state[]) {
    int numvib = 0;
    int n;
    for (n = 0; n < nmax; ++n) {
        numvib = numvib + mon_state[molecule_state[n] - 1].vib1 +
            mon_state[molecule_state[n] - 1].vib2;
    }

    return numvib;
}

double get_distance(int n1, int da1, int n2, int da2) {
    double distance = 0.0;
    int i = 9*n1 + 3*da1 - 12;
    int j = 9*n2 + 3*da2 - 12;
    distance = pow((mol1pos[i]-mol1pos[j]),2)
                  +pow((mol1pos[i+1]-mol1pos[j+1]),2)
                  +pow((mol1pos[i+2]-mol1pos[j+2]),2); 
    distance = sqrt(distance);

    return distance;
}

int get_charge(int state, int da){
    int charge = 0; 

    if ( state == es_z1 ){
        if ( da == leftdonor ){
            charge = 1;
        } else if ( da == acceptor ){
            charge = -1;
        }
    } else if( state == es_z2 ){
        if ( da == rightdonor ){
            charge = 1;
        } else if ( da == acceptor ){
            charge = -1;
        }
    }

    return charge; 
}


double factorial(int n) {
    double factorial_1 = 1.0;
    if (n < 0) {
        cout << "Factorial not calculatable for: " << n << endl;
        exit(0);
    }
    else {
        if (n != 0) {
            for (int i = 2; i <= n; i++) {
                factorial_1 = factorial_1 * i;
            }
        }
    }
    return factorial_1;
}

double volap(double lambda1, int vib1, double lambda2, int vib2) {
    double lambda = lambda2 - lambda1;
    double volap_1 = 0.0;
    for (int k = 0; k <= min(vib1, vib2); k++){
    volap_1 = volap_1 + pow((-1.0), (vib2 - k)) /
        (factorial(vib1 - k)*factorial(k)*
        factorial(vib2 - k))*
        pow(lambda, (vib1 + vib2 - 2 * k));
    }

    volap_1 = volap_1 * sqrt(1.0*factorial(vib1)*
        factorial(vib2))*exp(-1.0* pow(lambda, 2) / 2.0);

    if (volap_1 != volap_1) {
        cout << "Volap Error:: volap: " << volap_1 << endl;
        cout << "Aborting " << volap_1 << endl;
        exit(0);
    }
    return volap_1; 
}

// void print_matrix( char* desc, MKL_INT m, MKL_INT n, double* a, MKL_INT lda ) {
    // MKL_INT i, j;
    // printf( "\n %s\n", desc );
    // for( i = 0; i < m; ++i ) {
        // for( j = 0; j < n; ++j ) printf( " %6.2lf", a[i*lda+j] );
        // printf( "\n" );
    // }
// }

int main(int argc, char** argv) {

    time_t start=time(NULL), end;

    magma_print_environment();

    // GPU variables                 
    //const int           blockSize = 128;        // The number of threads to launch per block


    // ***              Variable Declaration            *** //
    // **************************************************** //

    //input file check
    if ( argc != 2 ){
        printf("Usage:\n"
               "\tInclude as the first argument either the name of an input file,  or a checkpoint\n"
               "\tfile with extension '.cpt' if restarting the calculation. No other arguments are\n"
               "\tallowed.\n");
        exit(EXIT_FAILURE);   
    }

    // retrieve and print info about gpu
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop,0);
    printf("\nGPU INFO:\n"
           "\tDevice name: %s\n"
           "\tMemory: %g gb\n",
           prop.name, prop.totalGlobalMem/(1.E9));
    
    // // register signal handler to take care of interruption and termination signals
    // signal( SIGINT,  signal_handler );
    // signal( SIGTERM, signal_handler );
    
    // read input file

    printf("\n>>> Setting parameters\n");
    
    std::ifstream file(argv[1],ifstream::in);
    if (!file)
    {
        cerr << "ERROR: unable to open input file: " << argv[1] << endl;
        exit(2);
    }
    string buff;
    string label;

    if (file.is_open()) {
        string line;
        int i, j, k;
        while (getline(file, line)) {
            // using printf() in all tests for consistency
            if (line[0] == '#') continue;

            for (i = 0; i <= line.length(); i++)
            {
                if (line[i] == ' ') {
                    break;
                }    
            }

            for (j = i; j <= line.length(); j++)
            {
                if (line[j] != ' ') {
                    break;
                }
            }

            for (k = j; k <= line.length(); k++)
            {
                if (line[k] == ' ') {
                    break;
                }
            }
            label = line.substr(0, i);
            buff = line.substr(j, k-j);

            if (label == "task_title") {
                task_title = buff;
                cout << "setting task_title to:" << task_title << endl;
                
            }
            else if (label == "nmax") {
                sscanf(buff.c_str(), "%d", &nmax);
                cout << "Setting nmax to: " << nmax << endl;
                
            }
            else if (label == "vibmax") {
                sscanf(buff.c_str(), "%d", &vibmax);
                cout << "Setting vibmax to: " << vibmax << endl;
                
            }
            else if (label == "sys_vibmax") {
                sscanf(buff.c_str(), "%d", &sys_vibmax);
                cout << "Setting sys_vibmax to: " << sys_vibmax << endl;
                
            }
            else if (label == "hw") {
                sscanf(buff.c_str(), "%lf", &hw);
                cout << "Setting vibration energy to: " << hw << endl;
                
            }
            else if (label == "calc_pl") {
                if (buff == ".true."){
                    calc_pl = true;
                    cout << "Will calculate all spectra " << endl;
                } else if(buff == ".false."){
                    calc_pl = false;
                    cout << "Will calculate only absorption " << endl;
                }
                
            }
            else if (label == "lorentzian") {
                if (buff == ".true."){
                    lorentzian = true;
                    cout << "Lineshape set to Lorentzian " << endl;
                } else if(buff == ".false."){
                    lorentzian = false;
                    cout << "Lineshape set to Gaussian " << endl;
                }
                
            }
            else if (label == "no_frenkel") {
                if (buff == ".true."){
                    no_frenkel = true;
                    cout << "no frenkel coupling will account " << endl;
                } else if(buff == ".false."){
                    no_frenkel = false;
                    cout << "frenkel coupling will account " << endl;
                }
                
            }
            else if (label == "periodic") {
                if (buff == ".true.") {
                    periodic = true;
                    cout << "periodic condition is on " << endl;
                }
                else if (buff == ".false.") {
                    periodic = false;
                    cout << "periodic condition is off " << endl;
                }
                
            }
            else if (label == "nearest_neighbor") {
                if (buff == ".true.") {
                    nearest_neighbor = true;
                    cout << "calc coupling from nearest neighbor " << endl;
                }
                else if (buff == ".false.") {
                    nearest_neighbor = false;
                    cout << "calc coupling from long range " << endl;
                }
                
            }
            else if (label == "lambda_n") {
                sscanf(buff.c_str(), "%lf", &lambda_n);
                cout << "Setting lambda_n to: " << lambda_n << endl;
                
            }
            else if (label == "lambda_z1") {
                sscanf(buff.c_str(), "%lf", &lambda_z1);
                cout << "Setting lambda_z1 to: " << lambda_z1 << endl;
                
            }
            else if (label == "lambda_z2") {
                sscanf(buff.c_str(), "%lf", &lambda_z2);
                cout << "Setting lambda_z2 to: " << lambda_z2 << endl;
                
            }
            else if (label == "agg_angle") {
                sscanf(buff.c_str(), "%lf", &agg_angle);
                cout << "Setting agg_angle to: " << agg_angle << endl;
                
            }
            else if (label == "dielectric") {
                sscanf(buff.c_str(), "%lf", &dielectric);
                cout << "Setting dielectric to: " << dielectric << endl;
                
            }
            else if (label == "abs_lw") {
                sscanf(buff.c_str(), "%lf", &abs_lw);
                cout << "Setting the abs linewidth to (cm-1): " << abs_lw << endl;
                
            }
            else if (label == "nz") {
                sscanf(buff.c_str(), "%lf", &nz);
                cout << "Setting zwitter energy to (cm-1): " << nz << endl;
                
            }
            else if (label == "tz") {
                sscanf(buff.c_str(), "%lf", &tz);
                cout << "Setting intra charge transfer to (cm-1): " << tz << endl;
                
            }
            else if (label == "spec_step") {
                sscanf(buff.c_str(), "%d", &spec_step);
                cout << "Setting spec_step to (cm-1): " << spec_step << endl;
                
            }
            else if (label == "spec_start_ab") {
                sscanf(buff.c_str(), "%lf", &spec_start_ab);
                cout << "Setting spec_start_ab to: " << spec_start_ab << endl;
                
            }
            else if (label == "spec_end_ab") {
                sscanf(buff.c_str(), "%lf", &spec_end_ab);
                cout << "Setting spec_end_ab to: " << spec_end_ab << endl;
                
            }
            else if (label == "spec_start_pl") {
                sscanf(buff.c_str(), "%lf", &spec_start_pl);
                cout << "Setting spec_start_pl to: " << spec_start_pl << endl;
                
            }
            else if (label == "spec_end_pl") {
                sscanf(buff.c_str(), "%lf", &spec_end_pl);
                cout << "Setting spec_end_pl to: " << spec_end_pl << endl;
                
            }
            else if (label == "xyz_file1") {
                xyz_file[0] = buff;
                cout << "Will read the xyz file: " << xyz_file[0] << endl;
                
            }
            else if (label == "xyz_file2") {
                xyz_file[1] = buff;
                cout << "Will read the xyz file: " << xyz_file[1] << endl;
                
            }
            // else if (label == "xyz_file3") {
            //     xyz_file[2] = buff;
            //     cout << "Will read the xyz file: " << xyz_file[2] << endl;
                
            // }
            else
                cout << "invalid label at line, " << label << buff << endl;
                //exit(2);
        }
        file.close();
    }
    
    
    // determine the number of blocks to launch on the gpu 
    // each thread parts of the matrix for diagonalization
    
    //set lambda variables
    es_lambda_z1[es_n - 1] = lambda_n;
    es_lambda_z1[es_z1 - 1] = lambda_z1;
    es_lambda_z1[es_z2 - 1] = lambda_n;
    es_lambda_z2[es_n - 1] = lambda_n;
    es_lambda_z2[es_z1 - 1] = lambda_n;
    es_lambda_z2[es_z2 - 1] = lambda_z2;

    if (vibmax == 0) {
        memset(es_lambda_z1, 0.0, sizeof(es_lambda_z1));
        memset(es_lambda_z2, 0.0, sizeof(es_lambda_z2));
    }

    int status = mkdir(task_title.c_str(), S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);
    chdir(task_title.c_str());
    
    //write the geometry file
    double angle = 55.0;
        angle =  angle/180.00*pi;
    double d = 4.300;
    double armlength = 5.9950;
    
    //molecule 1
    ofstream file2("mole1.dat");
    file2 << "8 1 0.0 0.0 0.0" << endl;
    file2 << "6 2 " << armlength*cos(-agg_angle/2)*sin(angle) <<" "<<
            armlength*sin(-agg_angle/2)*sin(angle) <<" "<< -armlength*cos(angle) << endl;
    file2 << "8 3 " << 2.0*armlength*cos(-agg_angle/2)*sin(angle) <<" "<<
            2.0*armlength*sin(-agg_angle/2)*sin(angle) << " 0.0" << endl;
    file2.close();
    
    //molecule 2
    ofstream file3("mole2.dat");
    file3 << "8 1 0.0 0.0 " << d << endl;
    file3 << "6 2 " << armlength*cos(agg_angle/2)*sin(angle) << " " <<
            armlength*sin(agg_angle/2)*sin(angle) <<" "<< d+armlength*cos(angle) << endl;
    file3 << "8 3 " << 2.0*armlength*cos(agg_angle/2)*sin(angle) <<" "<<
            2.0*armlength*sin(agg_angle/2)*sin(angle) <<" "<< d << endl;
    file3.close();

    int lattice_kount = 0; 
    for (int n1 = 1; n1 <= nmax; ++n1) {
        for (int anum1 = 1; anum1 <= anum; ++anum1) {
            for (int index = 1; index <= 3; ++index) {        //x,y,z
                lattice_kount += 1;
            }
        }
    }
    mol1pos = new double[lattice_kount];
    
    //build geometry
    int tmp, tmp1; 
    for (int n1 = 1; n1 <= nmax; ++n1) {
        string line;
        int anum2;

        printf("%s%s\n","Reading the xyz file: ", xyz_file[n1-1].c_str());
        ifstream file4(xyz_file[n1-1]);
        while (getline(file4, line)) {
            sscanf(line.c_str(), "%d %d", &tmp,&anum2);
            int i = 9*n1+3*anum2-12; 
            sscanf(line.c_str(), "%d %d %lf %lf %lf", &tmp,&tmp1, 
                &mol1pos[i],&mol1pos[i+1],&mol1pos[i+2]);
        }
    }
    // for (int n1 = 0; n1 < lattice_kount; n1++) {
    //     cout<< mol1pos[n1]<<endl;
    // }
    
    //monomer state index 
    for (int run = 1; run <= 2; run++) {
        mon_kount = 0; 
        for (int es_state = 1; es_state <= es_zmax; ++es_state) {
            for (int vib1 = 0; vib1 <= vibmax; ++vib1) {
                for (int vib2 = 0; vib2 <= vibmax; ++vib2) {
                    mon_kount += 1;
                    
                        if (run == 2){
                            mon_state[mon_kount - 1].es_state = es_state;
                            mon_state[mon_kount - 1].vib1 = vib1;
                            mon_state[mon_kount - 1].vib2 = vib2;
                        }
                }
            }
        }

        if (run == 1) {
            mon_state = new basis[mon_kount];
        }
    }
    cout << "monomer states are: " << mon_kount << endl;

    //system state index
    int *molecule_state = new int[nmax];

    for (int run = 1; run <= 2; run++) {
        sys_kount = 0;
        //cout << "system states are: " << endl;
        for (int sysnx = 0; sysnx < pow(mon_kount, nmax); ++sysnx) {
            for (int n = 0; n < nmax; ++n) {
                molecule_state[n] = get_mon_state(sysnx+1, n+1);
                //cout << "system states are: " << molecule_state[n] << endl;
            }
            
            if (get_numvib(molecule_state) > sys_vibmax) continue;
            sys_kount += 1;
            //cout << "system states" << get_numvib(molecule_state) << endl;
                if (run == 2) {
                     for (int n = 0; n < nmax; ++n) {
                        sys_state[sys_kount-1][n] = mon_state[molecule_state[n]-1];
                     }
                 }
        }
        if (run == 1) {
            sys_state = new basis*[sys_kount];
            for (int i = 0; i < sys_kount; ++i) 
                sys_state[i] = new basis[nmax];
        }
    }

    cout << "system states are: " << sys_kount << endl;
    // for (int sysnx = 0; sysnx < sys_kount; ++sysnx) {
        // for (int n = 0; n < nmax; ++n) {
            // cout << sys_state[sys_kount][n].es_state << endl;
            // cout << sys_state[sys_kount][n].vib1 << endl;
            // cout << sys_state[sys_kount][n].vib2 << endl;
        // }
    // }

    // coulomb coupling calculation
    coulomb_coupling = new double[sys_kount];
    for (int sysnx = 0; sysnx < sys_kount; ++sysnx) {
        coulomb_coupling[sysnx] = 0.0; 
        //cout << coulomb_coupling[sysnx] << endl;
    }
    
    const double eo         = 8.854187817*pow(10,-12); //(f/m)
    const double plancks    = 6.62606957*pow(10,-34);  //kg m**2 s**-2
    const double csol       = 2.99792458*pow(10,8);    //m s**-1
    const double aucharge_c = 1.602176565*pow(10,-19);  //c per au
    //const double au_debye   = 2.54175;       //au per debye
    double coeff = pow(aucharge_c,2)*(pow(10,10))/(4.0*pi*100.0*eo*plancks*csol)/dielectric; 
    
    if( nmax >=2 ){
        for (int sysnx = 1; sysnx <= sys_kount; ++sysnx){
            for (int n1 = 1; n1 < nmax; ++n1) {
                int state1 = sys_state[sysnx-1][n1-1].es_state; 

            for (int n2 = n1+1; n2 <= nmax; ++n2) {
                int state2 = sys_state[sysnx-1][n2-1].es_state; 
                    
                for (int da1 = 1; da1 <= anum; ++da1) {
                for (int da2 = 1; da2 <= anum; ++da2) {
                    // cout << get_distance(n1,da1,n2,da2) << endl;
                    
                    coulomb_coupling[sysnx-1]+= get_charge(state1, da1)
                            *get_charge(state2, da2)
                            /get_distance(n1,da1,n2,da2); 
                    // cout << coulomb_coupling[sysnx-1] << endl;

                }
                }
                
            }
            }
            coulomb_coupling[sysnx-1] = coulomb_coupling[sysnx-1] * coeff; 
            // cout << coulomb_coupling[sysnx-1] << endl;
        }
    }

    if(no_frenkel){
	    for (int sysnx = 0; sysnx < sys_kount; ++sysnx) {
	        coulomb_coupling[sysnx] = 0.0; 
	        //cout << coulomb_coupling[sysnx] << endl;
	    }	
    }
    
    // for (int sysnx = 0; sysnx < sys_kount; ++sysnx) {
        // cout << coulomb_coupling[sysnx] << endl;
    // }

    // transition dipole moment
    ux = new double[sys_kount];
    uy = new double[sys_kount];
    uz = new double[sys_kount];
    for (int sysnx = 0; sysnx < sys_kount; ++sysnx) {
        ux[sysnx] = 0.0; 
        uy[sysnx] = 0.0;
        uz[sysnx] = 0.0; 
    }
    

    for (int sysnx = 1; sysnx <= sys_kount; ++sysnx){
        for (int n1 = 1; n1 < nmax; ++n1) {
            int state1 = sys_state[sysnx-1][n1-1].es_state; 
            for (int da1 = 1; da1 <= anum; ++da1) {
                int i = 9*n1 + 3*da1 - 12; 
                ux[sysnx-1] += get_charge(state1, da1)* mol1pos[i];
                uy[sysnx-1] += get_charge(state1, da1)* mol1pos[i+1];
                uz[sysnx-1] += get_charge(state1, da1)* mol1pos[i+2];
            }
        }
    }


    //create hamiltonian 
    sys_h = new double[sys_kount*sys_kount];
    
    
    //initial matrix
    for (int state1 = 0; state1 < sys_kount; ++state1) {
        //d_w[state1] = 0.0; 
        for (int state2 = 0; state2 < sys_kount; ++state2) {
            sys_h[state1*sys_kount + state2] = 0.0;
        }
    }

    //on diagonal
    for (int state1 = 0; state1 < sys_kount; ++state1) {
        for (int n1 = 0; n1 < nmax; ++n1) {
            int state_a = sys_state[state1][n1].es_state;
            int vib_a1 = sys_state[state1][n1].vib1;
            int vib_a2 = sys_state[state1][n1].vib2;
            //cout << state_a << vib_a1 << vib_a2 << endl;
            //the electronic energy
            if (state_a == es_z1) {
                sys_h[state1*sys_kount + state1] += nz;
            }
            else if (state_a == es_z2) {
                sys_h[state1*sys_kount + state1] += nz;
            }
            //the vibration energy
            sys_h[state1*sys_kount + state1] += (vib_a1 + vib_a2)*hw; 
        }
        //Coulombic coupling for dimer 
        sys_h[state1*sys_kount + state1] += coulomb_coupling[state1]; 
        // cout << "count " << state1*sys_kount + state1<<sys_h[state1*sys_kount + state1] << endl;
    }

    //off diagonal
    for (int state1 = 0; state1 < sys_kount; ++state1) {
        for (int state2 = 0; state2 < sys_kount; ++state2) {
            if (state1 == state2) continue;
            //intramolecular CT
            int diff = 0;
            int diffn = 0;
            for (int n1 = 0; n1 < nmax; ++n1) {
                int state_a = sys_state[state1][n1].es_state;
                int vib_a1 = sys_state[state1][n1].vib1;
                int vib_a2 = sys_state[state1][n1].vib2;
                int state_b = sys_state[state2][n1].es_state;
                int vib_b1 = sys_state[state2][n1].vib1;
                int vib_b2 = sys_state[state2][n1].vib2;

                if (vib_a1 != vib_b1 || state_a != state_b || vib_a2 != vib_b2) {
                    diffn = n1;
                    diff += 1;
                }
                if (diff > 1) continue;
            }
            //at max, only one molecule can have a different configuration
            if (diff == 1) {
                int state_a = sys_state[state1][diffn].es_state;
                int vib_a1 = sys_state[state1][diffn].vib1;
                int vib_a2 = sys_state[state1][diffn].vib2;
                int state_b = sys_state[state2][diffn].es_state;
                int vib_b1 = sys_state[state2][diffn].vib1;
                int vib_b2 = sys_state[state2][diffn].vib2;

                if (state_a == es_n && state_b == es_z1) {
                    sys_h[state1*sys_kount + state2] = tz;
                }
                else if (state_a == es_n && state_b == es_z2) {
                    sys_h[state1*sys_kount + state2] = tz;
                }
                else if (state_a == es_z1 && state_b == es_n) {
                    sys_h[state1*sys_kount + state2] = tz;
                }
                else if (state_a == es_z2 && state_b == es_n) {
                    sys_h[state1*sys_kount + state2] = tz;
                }
                else {
                    sys_h[state1*sys_kount + state2] = 0.0;
                }

                //put vibration into
                double volapfact = volap(es_lambda_z1[state_a], vib_a1, 
                    es_lambda_z1[state_b], vib_b1)
                    *volap(es_lambda_z2[state_a], vib_a2, 
                        es_lambda_z2[state_b], vib_b2); 
                sys_h[state1*sys_kount + state2] = sys_h[state1*sys_kount + state2] * volapfact;
            }

        }
    }
    //print out the hamiltonian
    if (sys_kount < 100) {
        FILE* stream = fopen((task_title + "_H.dat").c_str(),"w");
        MKL_INT i,j; 
        
        //fprintf(stream, "Printing Matrix : \n");
        fprintf(stream, "\n %s\n", "Hamiltonian" );
        
        for( i = 0; i < sys_kount; ++i ) {
            for( j = 0; j < sys_kount; ++j ) fprintf(stream, " %6.2lf", sys_h[i*sys_kount+j]);
            fprintf(stream, "\n" ); 
        }
        //printf( "The algorithm failed to compute eigenvalues.\n" );
        fclose(stream);
        //if (outfile.is_open()) {
            //outfile << "Printing Matrix : \n";
            // for (int state1 = 0; state1 < sys_kount; ++state1) {
                // for (int state2 = 0; state2 < sys_kount; ++state2) {
                    // outfile << *(*(sys_h + state1) + state2) << " ";
                // }
                // outfile << endl;
            // }
            // outfile.close();
    }

    // magma variables for magma matrix print 
    magma_queue_t       queue;
    hipError_t         Cuerr;
    int                 Merr;
    
    // magma variables for dsyevd
    double         *d_sys_h;                                                       // the hamiltonian on the GPU
    double         aux_work[1];                                                    // To get optimal size of lwork
    magma_int_t         aux_iwork[1], info;                                             // To get optimal liwork, and return info
    magma_int_t         lwork, liwork;                                                  // Leading dim of kappa, sizes of work arrays
    magma_int_t         *iwork;                                                         // Work array
    double         *work;                                                          // Work array
    double         *w   ;                                                          // Eigenvalues
    double         *wA  ;                                                          // Work array
    
    //double         *d_w;                                                           // Eigenvalues on the GPU

    // Initialize magma math library and queue
    magma_init(); magma_queue_create( 0, &queue ); 
    
#define CHK_ERR     if (Cuerr != hipSuccess ) { printf(">>> ERROR on CUDA: %s.\n", hipGetErrorString(Cuerr)); exit(EXIT_FAILURE);}
#define MALLOC_ERR  { printf(">>> ERROR on CPU: out of memory.\n"); exit(EXIT_FAILURE);}
#define CHK_MERR    if (Merr != MAGMA_SUCCESS ) { printf(">>> ERROR on MAGMA: %s.\n", magma_strerror(Merr)); exit(EXIT_FAILURE);}
    int SSYEVD_ALLOC_FLAG = 1;     // flag whether to allocate ssyevr arrays -- it is turned off after they are allocated
    
    magma_int_t sys_kount2 = (magma_int_t) sys_kount*sys_kount; 
    
    //allocate memory in GPU
    Cuerr = hipMalloc (&d_sys_h,sys_kount2*sizeof(double)); CHK_ERR;
    


    //begin the diagonalization
    hipMemcpy( d_sys_h, sys_h, sys_kount2* sizeof(double), hipMemcpyHostToDevice );

    // if the first time, query for optimal workspace dimensions
    if ( SSYEVD_ALLOC_FLAG )
    {   
        magma_dsyevd_gpu( MagmaVec, MagmaUpper, (magma_int_t) sys_kount, NULL, (magma_int_t) sys_kount, 
            NULL, NULL, (magma_int_t) sys_kount, aux_work, -1, aux_iwork, -1, &info );
        
        lwork  = (magma_int_t) MAGMA_D_REAL( aux_work[0] );
        liwork  = aux_iwork[0];

        // allocate work arrays, eigenvalues and other stuff
        
        Merr = magma_imalloc_cpu   ( &iwork, liwork ); CHK_MERR; 
        Merr = magma_dmalloc_pinned( &wA , sys_kount2 ) ; CHK_MERR;
        Merr = magma_dmalloc_cpu   ( &w,     sys_kount ); CHK_MERR; 
        Merr = magma_dmalloc_pinned( &work , lwork  ); CHK_MERR;

        SSYEVD_ALLOC_FLAG = 0;      // is allocated here, so we won't need to do it again
        //cout<< "Hamiltonian" <<endl;

        // get info about space needed for diagonalization
        size_t freem, total;
        hipMemGetInfo( &freem, &total );
        printf("\n>>> hipMemGetInfo returned\n"
               "\tfree:  %g gb\n"
               "\ttotal: %g gb\n", (double) freem/(1E9), (double) total/(1E9));
        printf(">>> %g gb needed by diagonalization routine.\n", (double) (lwork * (double) sizeof(double)/(1E9)));
    }

    magma_dsyevd_gpu( MagmaVec, MagmaUpper, (magma_int_t) sys_kount, d_sys_h, (magma_int_t) sys_kount,
        w, wA, (magma_int_t) sys_kount, work, lwork, iwork, liwork, &info );

    if ( info != 0 ){ printf("ERROR: magma_dsyevd_gpu returned info %lld.\n", info ); exit(EXIT_FAILURE);}
                
    // copy eigenvalues to device memory
    //hipMemcpy( d_w    , w    , sys_kount*sizeof(double), hipMemcpyDeviceToHost );
    hipMemcpy( sys_h, d_sys_h, sys_kount2* sizeof(double), hipMemcpyDeviceToHost );
    
    //test
    // magma_dprint_gpu((magma_int_t) sys_kount,(magma_int_t) sys_kount,
    // d_sys_h,(magma_int_t) sys_kount,queue); 	
    
    //magma_dprint( (magma_int_t) sys_kount, (magma_int_t) sys_kount, sys_h, (magma_int_t) sys_kount); 

    //transpose
    double temp; 
    int i,j;
    for( i = 0; i < sys_kount; ++i ) {
        for( j = i; j < sys_kount; ++j ) {
            temp = sys_h[i*sys_kount+j];
            sys_h[i*sys_kount+j] = sys_h[j*sys_kount+i]; 
            sys_h[j*sys_kount+i] = temp ; 
        }
    }

    //print eigenvalue & eigenvectors
    if (sys_kount < 100) {
        FILE* stream = fopen((task_title + "_evec.dat").c_str(),"w");
        MKL_INT i,j; 
        
        //fprintf(stream, "Printing Matrix : \n");
        fprintf(stream, "\n %s\n", "Eigenvalues" );
        for( j = 0; j < sys_kount; ++j ) fprintf(stream, " %6.2lf", w[j] );
        fprintf(stream, "\n" );
        
        fprintf(stream, "\n %s\n", "Eigenvectors (stored columnwise)" );
        
        for( i = 0; i < sys_kount; ++i ) {
            for( j = 0; j < sys_kount; ++j ) fprintf(stream, " %6.4lf", sys_h[i*sys_kount+j]);
            fprintf( stream, "\n" );
        }
    
        fclose(stream);
    }
    
    //absorption spectrum
    //absorption oscilator strength
    //double lineshape; 
    ab_osc_x = new double[sys_kount-1];
    ab_osc_y = new double[sys_kount-1];
    ab_osc_z = new double[sys_kount-1];
    ab_sys_eval = new double[sys_kount-1];

    for (int sysnx = 0; sysnx < sys_kount-1; ++sysnx) {
        ab_osc_x[sysnx] = 0.0; 
        ab_osc_y[sysnx] = 0.0;
        ab_osc_z[sysnx] = 0.0; 
    }
    
    int state; 
    
    for (state = 1; state < sys_kount; ++state){
        ab_sys_eval[state-1] = *(w + state) - *(w); 
        for(int hx = 0; hx < sys_kount; ++hx ){
            ab_osc_x[state-1] += ux[hx] * sys_h[hx*sys_kount+state] * sys_h[hx*sys_kount];
            ab_osc_y[state-1] += uy[hx] * sys_h[hx*sys_kount+state] * sys_h[hx*sys_kount];
            ab_osc_z[state-1] += uz[hx] * sys_h[hx*sys_kount+state] * sys_h[hx*sys_kount];
        }
        ab_osc_x[state-1] = pow(ab_osc_x[state-1],2);
        ab_osc_y[state-1] = pow(ab_osc_y[state-1],2);
        ab_osc_z[state-1] = pow(ab_osc_z[state-1],2);
    }
    
     //find the highest oscillator strength
    double temp_x = 0;
    double temp_y = 0;
    //double temp_z = 0;
    
    int pl_start_x, pl_start_y; 
    
    for (state = 0; state < sys_kount-1; state++){
        if (ab_osc_x[state]>temp_x){
            temp_x = ab_osc_x[state];
            pl_start_x = state;
        }
        if (ab_osc_y[state]>temp_y){
            temp_y = ab_osc_y[state];
            pl_start_y = state;
        }
    }

        
    pl_start = min(pl_start_x,pl_start_y); 
    printf( "lowest excited state is: %d \n" , pl_start);
    
    //absorption spectrum
    ab_x = new double[spec_step];
    ab_y = new double[spec_step];
    ab_z = new double[spec_step];
    for (int spec_point = 0; spec_point < spec_step; ++spec_point) {
        ab_x[spec_point] = 0.0; 
        ab_y[spec_point] = 0.0;
        ab_z[spec_point] = 0.0; 
    }
    
    for (int spec_point = 0; spec_point < spec_step; spec_point++){
        double energy = spec_start_ab + (spec_end_ab - spec_start_ab)/spec_step*(spec_point+1); 
        for(int state = 0; state < sys_kount-1; state++ ){
            double tran_e = ab_sys_eval[state];
            double lineshape; 
            if (lorentzian){
                lineshape = abs_lw/(pow((energy-tran_e),2)+pow(abs_lw,2))/pi;
            }
            else{
                lineshape = exp(-(pow((energy - tran_e)/abs_lw,2)));
            }
            
            if ( abs_freq_dep ){
                ab_x[spec_point] += lineshape * ab_osc_x[state] * tran_e/pow(10.0,4);
                ab_y[spec_point] += lineshape * ab_osc_y[state] * tran_e/pow(10.0,4);
                ab_z[spec_point] += lineshape * ab_osc_z[state] * tran_e/pow(10.0,4);
            }
            else{
                ab_x[spec_point] += lineshape * ab_osc_x[state];
                ab_y[spec_point] += lineshape * ab_osc_y[state];
                ab_z[spec_point] += lineshape * ab_osc_z[state];
            }
        }
    }
    
    //print absorption spectrum
    FILE* stream = fopen((task_title + "_ab.dat").c_str(),"w");
    
        // fprintf(stream, "Printing Matrix : \n");
        fprintf(stream, "%s\n", "Energy A(g(w))" );
        fprintf(stream, "%s\n", "Energy System" );
        fprintf(stream, "%s\n\n", "cm +(-1) a.u." );
        
        for(int spec_point = 0; spec_point < spec_step; ++spec_point ){
            double energy = spec_start_ab + (spec_end_ab - spec_start_ab)/spec_step*(spec_point+1);
            fprintf(stream, " %lf %lf %lf %lf %lf\n", energy, ab_x[spec_point]+ab_y[spec_point]+ab_z[spec_point],
                    ab_x[spec_point], ab_y[spec_point], ab_z[spec_point]);
        }

    fclose(stream);
    
    // pl spectrum
    // pl oscilator strength
    pl_osc_x = new double[pl_start-1];
    pl_osc_y = new double[pl_start-1];
    pl_osc_z = new double[pl_start-1];
    pl_sys_eval = new double[pl_start-1];
    for (int sysnx = 0; sysnx < pl_start-1; ++sysnx) {
        pl_osc_x[sysnx] = 0.0; 
        pl_osc_y[sysnx] = 0.0;
        pl_osc_z[sysnx] = 0.0; 
    }
    
    for (int state = 1; state < pl_start; ++state){
        pl_sys_eval[state-1] = *(w + pl_start) - *(w + state - 1); 
        for(int hx = 0; hx < sys_kount; hx++ ){
            pl_osc_x[state-1] += ux[hx] * sys_h[hx*sys_kount+state-1] * sys_h[hx*sys_kount+pl_start];
            pl_osc_y[state-1] += uy[hx] * sys_h[hx*sys_kount+state-1] * sys_h[hx*sys_kount+pl_start];
            pl_osc_z[state-1] += uz[hx] * sys_h[hx*sys_kount+state-1] * sys_h[hx*sys_kount+pl_start];
        }
        pl_osc_x[state-1] = pl_osc_x[state-1] * pl_osc_x[state-1];
        pl_osc_y[state-1] = pl_osc_y[state-1] * pl_osc_y[state-1];
        pl_osc_z[state-1] = pl_osc_z[state-1] * pl_osc_z[state-1];
    }
    
    //pl spectrum
    pl_x = new double[spec_step];
    pl_y = new double[spec_step];
    pl_z = new double[spec_step];
    for (int spec_point = 0; spec_point < spec_step; ++spec_point) {
        pl_x[spec_point] = 0.0; 
        pl_y[spec_point] = 0.0;
        pl_z[spec_point] = 0.0; 
    }
    
    for (int spec_point = 0; spec_point < spec_step; ++spec_point){
        double energy = spec_start_pl + (spec_end_pl - spec_start_pl)/spec_step*(spec_point+1);
        for(int state = 0; state < pl_start-1; ++state ){
            double tran_e = pl_sys_eval[state]; 
            double lineshape; 
            if (lorentzian){
                lineshape = abs_lw/(pow((energy-tran_e),2)+pow(abs_lw,2))/pi; 
            }
            else{
                lineshape = exp(-pow((energy - tran_e)/abs_lw,2)); 
            }
            
            pl_x[spec_point] += lineshape * pl_osc_x[state];
            pl_y[spec_point] += lineshape * pl_osc_y[state];
            pl_z[spec_point] += lineshape * pl_osc_z[state];

            if ( abs_freq_dep ){
                pl_x[spec_point] += lineshape * pl_osc_x[state] * pow(tran_e,3)/pow(10.0,12);
                pl_y[spec_point] += lineshape * pl_osc_y[state] * pow(tran_e,3)/pow(10.0,12);
                pl_z[spec_point] += lineshape * pl_osc_z[state] * pow(tran_e,3)/pow(10.0,12);
            }
            else{
                pl_x[spec_point] += lineshape * pl_osc_x[state];
                pl_y[spec_point] += lineshape * pl_osc_y[state];
                pl_z[spec_point] += lineshape * pl_osc_z[state];
            }
        }
    }
    
    //print absorption spectrum
    FILE* stream1 = fopen((task_title + "_pl.dat").c_str(),"w");
    
        // fprintf(stream, "Printing Matrix : \n");
        fprintf(stream1, "%s\n", "Energy F(g(w))" );
        fprintf(stream1, "%s\n", "Energy System" );
        fprintf(stream1, "%s\n\n", "cm +(-1) a.u." );
        
        for(int spec_point = 0; spec_point < spec_step; ++spec_point ){
            double energy = spec_start_pl + (spec_end_pl - spec_start_pl)/spec_step*(spec_point+1);
            fprintf(stream1, " %lf %lf %lf %lf %lf\n", energy, pl_x[spec_point]+pl_y[spec_point]+pl_z[spec_point],
                    pl_x[spec_point], pl_y[spec_point], pl_z[spec_point]);
        }

    fclose(stream1);
    
    //paremeters out
    ofstream file1(task_title + "_para.csv");
    file1 << "parameter, value" << endl;
    file1 << "@@@@@@@@@@@@@@@@" << endl;
    file1 << "task title, " << task_title << endl;
    file1 << "es_zmax, " << es_zmax << endl;
    file1 << "nmax, " << nmax << endl;
    file1 << "vibrational energy (cm-1), " << hw << endl;
    file1 << "Ion pair energy (cm-1), " << nz << endl;
    file1 << "Intramolecule charge transfer (cm-1), " << tz << endl;
    file1 << "vibmax, " << vibmax << endl;
    file1 << "sys_vibmax, " << sys_vibmax << endl;
    file1 << "lambda_n, " << lambda_n << endl;
    file1 << "lambda_z1, " << lambda_z1 << endl;
    file1 << "lambda_z2, " << lambda_z2 << endl;
    file1 << "monomer kount, " << mon_kount << endl;
    file1 << "system kount, " << sys_kount << endl;
    file1 << "abs linewidth (cm-1), " << abs_lw  << endl;
    file1 << "dielectric, " << dielectric  << endl;
    file1 << "no_frenkel, " << no_frenkel  << endl;
    file1 << "periodic, " << periodic  << endl;    
    file1.close();
    

    end = time(NULL);
    printf("\n>>> Done with the calculation in %f seconds.\n", difftime(end,start));

    
    //free memory 
    // delete[] eigenvector;

    // for (int i = 0; i < sys_kount; i++)
    //     delete[] sys_h[i];
    delete[] sys_h;

    
    delete[] mon_state; 
    for (int i = 0; i < sys_kount; i++)
        delete[] sys_state[i];
    delete[] sys_state;


    delete[] ab_osc_x;
    delete[] ab_osc_y;
    delete[] ab_osc_z;      
    delete[] ab_x;    
    delete[] ab_y;    
    delete[] ab_z; 

    delete[] ux;    
    delete[] uy;    
    delete[] uz;    

    delete[] pl_osc_x;
    delete[] pl_osc_y;
    delete[] pl_osc_z;      
    delete[] pl_x;    
    delete[] pl_y;    
    delete[] pl_z; 

    delete[] coulomb_coupling;
    delete[] mol1pos;

    

    // free memory on the CPU and GPU and finalize magma library
    
    hipFree(d_sys_h);
    
    //hipFree(d_w);
    if ( SSYEVD_ALLOC_FLAG == 0 )
    {
        free(w);
        free(iwork);
        magma_free_pinned( work );
        magma_free_pinned( wA );
    }
    
    exit(0);
    
    // auto stop = high_resolution_clock::now(); 
    // auto duration = duration_cast<microseconds>(stop - start);
    // cout << duration.count() << endl; 

    // t2=clock();
    // float diff ((float)t2-(float)t1);
    // cout<<diff<<endl;

    // hipEventRecord(stop, 0);
    // hipEventSynchronize (stop) ;
    // hipEventElapsedTime(&elapsed, start, stop) ;
    // hipEventDestroy(start);
    // hipEventDestroy(stop);


    // final call to finalize magma math library
    magma_finalize();

    return 0;

}
